#include "hip/hip_runtime.h"
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>

#define IDX(iy, ix, nx) ((iy)*(nx)+(ix))
#define BLOCK 16

// Petite fonction pour afficher la grille finale (depuis le host)
void printState(const double* a, int nx, int ny) {
    for (int iy = 0; iy < ny; iy++) {
        for (int ix = 0; ix < nx; ix++)
            printf("%.6f ", a[iy * nx + ix]);
        printf("\n");
    }
    printf("\n");
}

// J'initialise les bords à 1 et tout le reste à 0 (sur le device)
__global__
void init_boundaries(double *a, double *a_new, int nx, int ghost_ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= nx || iy >= ghost_ny) return;
    if (ix == 0 || ix == nx-1) {
        a[IDX(iy,ix,nx)]     = 1.0;
        a_new[IDX(iy,ix,nx)] = 1.0;
    } else {
        a[IDX(iy,ix,nx)]     = 0.0;
        a_new[IDX(iy,ix,nx)] = 0.0;
    }
}

// Kernel Jacobi classique : on calcule la nouvelle grille pour tout le sous-domaine (hors bords physiques)
__global__
void jacobi_kernel_full(const double *a, double *a_new, int nx, int ghost_ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix < 1 || ix >= nx-1 || iy < 1 || iy >= ghost_ny-1) return;
    int idx = IDX(iy,ix,nx);
    a_new[idx] = 0.25 * (
        a[idx - nx] +
        a[idx + nx] +
        a[idx - 1 ] +
        a[idx + 1 ]
    );
}

// Pour choper le max des différences entre deux vecteurs (pour la convergence)
struct max_abs_diff {
    __host__ __device__
    double operator()(const thrust::tuple<double,double>& t) const {
        return fabs(thrust::get<0>(t) - thrust::get<1>(t));
    }
};

// Petite fonction utilitaire pour swap deux pointeurs device
void swap_ptrs(double **p, double **q) {
    double *t = *p; *p = *q; *q = t;
}

int main(int argc, char **argv) {
    // 1) Chrono global pour mesurer TOUT le programme (alloc, init, calcul, print...)
    clock_t global_start = clock();
    double t_nvshmem_prep_start = MPI_Wtime();

    MPI_Init(&argc, &argv);
    int rank, size;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    // Attribution du GPU local pour chaque processus MPI sur le nœud (multinode/multigpu)
    int local_rank = 0;
    if (size > 1) {
        MPI_Comm local_comm;
        MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL, &local_comm);
        MPI_Comm_rank(local_comm, &local_rank);
        hipSetDevice(local_rank);
        hipFree(0);
        MPI_Comm_free(&local_comm);
    } else {
        hipSetDevice(0);
        hipFree(0);
    }

    // Lecture des arguments
    if (argc != 6) {
        if (!rank) fprintf(stderr, "Usage: %s nx ny max_iter tol print_flag\n", argv[0]);
        MPI_Finalize();
        return 1;
    }
    int nx = atoi(argv[1]);
    int ny = atoi(argv[2]);
    int max_iter = atoi(argv[3]);
    double tol = atof(argv[4]);
    int print_flag = atoi(argv[5]);

    // Je découpe la grille pour répartir le boulot de façon équilibrée (sous-domaines)
    int work_ny = ny - 2;
    int chunk_size_low  = work_ny / size;
    int chunk_size_high = chunk_size_low + 1;
    int num_ranks_low   = size * chunk_size_low + size - work_ny;
    int local_ny, iy_start_global;
    if (rank < num_ranks_low) {
        local_ny = chunk_size_low;
        iy_start_global = rank * chunk_size_low + 1;
    } else {
        local_ny = chunk_size_high;
        iy_start_global = num_ranks_low * chunk_size_low + (rank - num_ranks_low) * chunk_size_high + 1;
    }
    int ghost_ny = local_ny + 2;

    // Je construis la taille de ghost_ny pour chaque PE (utile si partitionnement non uniforme)
    int* ghost_nys = (int*)malloc(size * sizeof(int));
    int* local_nys = (int*)malloc(size * sizeof(int));
    MPI_Allgather(&local_ny, 1, MPI_INT, local_nys, 1, MPI_INT, MPI_COMM_WORLD);
    for(int i=0; i<size; i++) ghost_nys[i] = local_nys[i] + 2;

    // Je prépare le heap symétrique NVSHMEM (mémoire identique partout)
    int max_ghost_ny;
    MPI_Allreduce(&ghost_ny, &max_ghost_ny, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD);
    size_t bytes = max_ghost_ny * nx * sizeof(double);
    size_t required_heap = 2 * bytes * 1.1;
    if (!getenv("NVSHMEM_SYMMETRIC_SIZE")) {
        char buf[64];
        sprintf(buf, "%zu", required_heap);
        setenv("NVSHMEM_SYMMETRIC_SIZE", buf, 1);
    }

    // Préparation de la structure d'init pour NVSHMEM (mais on n'initialise pas tout de suite)
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr_t attr;
    attr.mpi_comm = &mpi_comm;
    double t_nvshmem_prep_end = MPI_Wtime();

    // Chrono juste pour l'init NVSHMEM
    double t_nvshmem_init_start = MPI_Wtime();
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    double t_nvshmem_init_end = MPI_Wtime();

    nvshmem_barrier_all();
    double t_after_nvshmem_init = MPI_Wtime();

    // Allocation mémoire sur heap symétrique NVSHMEM (même si size=1 pour être uniforme)
    double *d_a     = (double*)nvshmem_malloc(bytes);
    double *d_a_new = (double*)nvshmem_malloc(bytes);

    // Initialisation des bords sur le device
    dim3 block(BLOCK, BLOCK);
    dim3 grid((nx + BLOCK-1)/BLOCK, (ghost_ny + BLOCK-1)/BLOCK);
    init_boundaries<<<grid, block>>>(d_a, d_a_new, nx, ghost_ny);
    hipDeviceSynchronize();

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Barrière pour tout le monde avant de commencer le vrai calcul
    nvshmem_barrier_all();
    double t_calc0 = MPI_Wtime();

    double error = tol + 1.0;
    int iter = 0;

    // === BLOC PRINCIPAL JACOBI ===
    while (error > tol && iter < max_iter) {
        // 1) Calcul du Jacobi sur tout le sous-domaine (hors bords)
        jacobi_kernel_full<<<grid, block, 0, stream>>>(d_a, d_a_new, nx, ghost_ny);
        hipStreamSynchronize(stream);

        // 2) Calcul du critère d'arrêt : max des différences absolues (avec thrust)
        thrust::device_ptr<double> ptr_new(d_a_new);
        thrust::device_ptr<double> ptr_old(d_a);
        error = thrust::transform_reduce(
            thrust::make_zip_iterator(thrust::make_tuple(ptr_new, ptr_old)),
            thrust::make_zip_iterator(thrust::make_tuple(ptr_new + ghost_ny*nx, ptr_old + ghost_ny*nx)),
            max_abs_diff(),
            0.0,
            thrust::maximum<double>());

        // 3) MPI_Allreduce pour avoir la norme max globale (même en séquentiel, pour la cohérence des timings)
        MPI_Allreduce(MPI_IN_PLACE, &error, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

        // 4) Swap des pointeurs pour la prochaine itération
        swap_ptrs(&d_a, &d_a_new);
        iter++;
    }

    nvshmem_barrier_all();
    double t_calc1 = MPI_Wtime();

    // On synchronise tout le monde avant les mesures "setup+calcul+affichage"
    nvshmem_barrier_all();
    double t_after_all = MPI_Wtime();

    // Affichage des différents temps sur le rang 0 uniquement
    if (rank == 0) {
        printf("[NVSHMEM-baseline-single-rank] Converged in %d iterations | Error: %.2e\n", iter, error);
        printf("Temps de calcul Jacobi (seulement boucle)         : %.6fs\n", t_calc1 - t_calc0);
        printf("Temps setup+calcul+affichage (après init SHMEM)   : %.6fs\n", t_after_all - t_after_nvshmem_init);
        printf("Temps d'init avant NVSHMEM                        : %.6fs\n", t_nvshmem_prep_end - t_nvshmem_prep_start);
        printf("Temps d'init NVSHMEM                              : %.6fs\n", t_nvshmem_init_end - t_nvshmem_init_start);
        double global_time = (double)(clock() - global_start) / CLOCKS_PER_SEC;
        printf("Temps total du programme (tout compris)           : %.6fs\n", global_time);
    }

    // Affichage de la solution finale (si demandé)
    if (print_flag) {
        double* local_result = (double*)malloc(local_ny * nx * sizeof(double));
        hipMemcpy(local_result, d_a + IDX(1,0,nx), local_ny * nx * sizeof(double), hipMemcpyDeviceToHost);

        if (rank == 0) {
            double* full = (double*)malloc(ny * nx * sizeof(double));
            memcpy(full + iy_start_global * nx, local_result, local_ny * nx * sizeof(double));
            for (int r = 1; r < size; r++) {
                int r_ny, r_offset;
                if (r < num_ranks_low) {
                    r_ny = chunk_size_low;
                    r_offset = r * chunk_size_low + 1;
                } else {
                    r_ny = chunk_size_high;
                    r_offset = num_ranks_low * chunk_size_low + (r - num_ranks_low) * chunk_size_high + 1;
                }
                MPI_Recv(full + r_offset * nx, r_ny * nx, MPI_DOUBLE, r, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            }
            // On recopie les bords (y=0 et y=ny-1)
            memcpy(full, full + nx, nx * sizeof(double));
            memcpy(full + (ny-1)*nx, full + (ny-2)*nx, nx * sizeof(double));
            printf("État final :\n");
            printState(full, nx, ny);
            free(full);
        } else {
            MPI_Send(local_result, local_ny * nx, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
        }
        free(local_result);
    }

    // Libération de la mémoire
    if(d_a) nvshmem_free(d_a);
    if(d_a_new) nvshmem_free(d_a_new);
    hipStreamDestroy(stream);
    nvshmem_finalize();
    MPI_Finalize();
    free(ghost_nys);
    free(local_nys);
    return 0;
}
